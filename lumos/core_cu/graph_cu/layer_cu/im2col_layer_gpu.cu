#include "im2col_layer_gpu.h"

void forward_im2col_layer_gpu(Layer l, int num)
{
    hipMemcpy(l.output, l.input, num*l.outputs*sizeof(float), hipMemcpyDeviceToDevice);
    float output_cpu[l.outputs];
    hipMemcpy(output_cpu, l.output, l.outputs*sizeof(float), hipMemcpyDeviceToHost);
    // printf("\n\n\n");
    // for (int i = 0; i < l.outputs; ++i){
    //     printf("%f ", output_cpu[i]);
    // }
    // printf("\n\n\n");
}

void backward_im2col_layer_gpu(Layer l, float rate, int num, float *n_delta)
{
    hipMemcpy(l.delta, n_delta, num*l.inputs*sizeof(float), hipMemcpyDeviceToDevice);
}
