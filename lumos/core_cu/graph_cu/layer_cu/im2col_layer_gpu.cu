#include "im2col_layer_gpu.h"

void init_im2col_layer_gpu(Layer *l, int w, int h, int c, int subdivision)
{
    l->input_h = h,
    l->input_w = w;
    l->input_c = c;
    l->inputs = l->input_h * l->input_w * l->input_c;

    l->output_h = 1;
    l->output_w = 1;
    l->output_c = l->inputs;
    l->outputs = l->inputs;

    l->workspace_size = 0;
    hipMalloc((void**)&l->output, subdivision*l->outputs*sizeof(float));
    hipMalloc((void**)&l->delta, subdivision*l->inputs*sizeof(float));

    fprintf(stderr, "Im2col          Layer    %3d*%3d*%3d ==> %3d*%3d*%3d\n",
            l->input_w, l->input_h, l->input_c, l->output_w, l->output_h, l->output_c);
}

void forward_im2col_layer_gpu(Layer l, int num)
{
    hipMemcpy(l.output, l.input, num*l.outputs*sizeof(float), hipMemcpyDeviceToDevice);
}

void backward_im2col_layer_gpu(Layer l, float rate, int num, float *n_delta)
{
    hipMemcpy(l.delta, n_delta, num*l.inputs*sizeof(float), hipMemcpyDeviceToDevice);
}

void free_im2col_layer_gpu(Layer l)
{
    hipFree(l.output);
    hipFree(l.delta);
}