#include "im2col_layer_gpu.h"

void init_im2col_layer_gpu(Layer *l, int w, int h, int c)
{
    l->input_h = h,
    l->input_w = w;
    l->input_c = c;
    l->inputs = l->input_h * l->input_w * l->input_c;

    l->output_h = 1;
    l->output_w = 1;
    l->output_c = l->inputs;
    l->outputs = l->inputs;
    l->workspace_size = 0;

    l->forward = forward_im2col_layer_gpu;
    l->backward = backward_im2col_layer_gpu;

    hipMalloc((void**)&l->output, l->outputs*l->subdivision*sizeof(float));
    hipMalloc((void**)&l->delta, l->inputs*l->subdivision*sizeof(float));

    fprintf(stderr, "Im2col          Layer    %3d*%3d*%3d ==> %3d*%3d*%3d\n",
            l->input_w, l->input_h, l->input_c, l->output_w, l->output_h, l->output_c);
}

void forward_im2col_layer_gpu(Layer l, int num)
{
    hipMemcpy(l.output, l.input, num*l.outputs*sizeof(float), hipMemcpyDeviceToDevice);
}

void backward_im2col_layer_gpu(Layer l, float rate, int num)
{
    hipMemcpy(l.delta, l.n_delta, num*l.inputs*sizeof(float), hipMemcpyDeviceToDevice);
}
