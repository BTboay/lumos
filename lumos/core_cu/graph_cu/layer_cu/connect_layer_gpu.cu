#include "hip/hip_runtime.h"
#include "connect_layer_gpu.h"

void init_connect_layer_gpu(Layer *l, int w, int h, int c, int subdivision)
{
    l->input_h = h;
    l->input_w = w;
    l->input_c = c;
    l->inputs = l->input_h * l->input_w * l->input_c;

    l->output_h = 1;
    l->output_w = 1;
    l->output_c = l->ksize;
    l->outputs = l->output_h * l->output_w * l->output_c;

    l->workspace_size = l->inputs * l->outputs;

    hipMalloc((void**)&l->output, subdivision*l->outputs*sizeof(float));
    hipMalloc((void**)&l->delta, subdivision*l->inputs*sizeof(float));
    hipMalloc((void**)&l->kernel_weights, l->inputs*l->outputs*sizeof(float));
    hipMalloc((void**)&l->update_kernel_weights, l->inputs*l->outputs*sizeof(float));
    if (l->bias){
        hipMalloc((void**)&l->bias_weights, l->outputs*sizeof(float));
        hipMalloc((void**)&l->update_bias_weights, l->outputs*sizeof(float));
    }

    fprintf(stderr, "Connect         Layer    %3d*%3d*%3d ==> %3d*%3d*%3d\n",
            l->input_w, l->input_h, l->input_c, l->output_w, l->output_h, l->output_c);
}

void weightinit_connect_layer_gpu(Layer l, FILE *fp)
{
    if (fp){
        float *kernel_weights = (float*)calloc(l.inputs*l.outputs, sizeof(float));
        fread(kernel_weights, sizeof(float), l.outputs*l.inputs, fp);
        hipMemcpy(l.kernel_weights, kernel_weights, l.inputs*l.outputs*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(l.update_kernel_weights, kernel_weights, l.inputs*l.outputs*sizeof(float), hipMemcpyHostToDevice);
        free(kernel_weights);
        if (l.bias){
            float *bias_weights = (float*)calloc(l.outputs, sizeof(float));
            fread(bias_weights, sizeof(float), l.outputs, fp);
            hipMemcpy(l.bias_weights, bias_weights, l.outputs*sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(l.update_bias_weights, bias_weights, l.outputs*sizeof(float), hipMemcpyHostToDevice);
            free(bias_weights);
        }
        return;
    }
    float *kernel_weights = (float*)calloc(l.inputs*l.outputs, sizeof(float));
    float scale = sqrt((float)2 / l.inputs);
    for (int i = 0; i < l.inputs*l.outputs; ++i){
        kernel_weights[i] = scale*rand_uniform(-1, 1);
    }
    if (l.bias){
        float *bias_weights = (float*)calloc(l.outputs, sizeof(float));
        fill_cpu(bias_weights, l.outputs, 0.001, 1);
        hipMemcpy(l.bias_weights, bias_weights, l.outputs*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(l.update_bias_weights, bias_weights, l.outputs*sizeof(float), hipMemcpyHostToDevice);
        free(bias_weights);
    }
    hipMemcpy(l.kernel_weights, kernel_weights, l.inputs*l.outputs*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(l.update_kernel_weights, kernel_weights, l.inputs*l.outputs*sizeof(float), hipMemcpyHostToDevice);
    free(kernel_weights);
}

void forward_connect_layer_gpu(Layer l, int num)
{
    for (int i = 0; i < num; ++i){
        int offset_i = i * l.inputs;
        int offset_o = i * l.outputs;
        float *input = l.input + offset_i;
        float *output = l.output + offset_o;
        gemm_gpu(0, 0, l.outputs, l.inputs, l.inputs, 1,
             1, l.kernel_weights, input, output);
        if (l.bias){
            add_bias_gpu(output, l.bias_weights, l.ksize, 1);
        }
        activate_list_gpu(output, l.outputs, l.active);
    }
}

void backward_connect_layer_gpu(Layer l, float rate, int num, float *n_delta)
{
    for (int i = 0; i < num; ++i){
        int offset_i = i * l.inputs;
        int offset_o = i * l.outputs;
        float *output = l.output + offset_o;
        float *delta_l = l.delta + offset_i;
        float *delta_n = n_delta + offset_o;
        gradient_list_gpu(output, l.outputs, l.active);
        matrix_multiply_gpu(delta_n, output, l.outputs, delta_n);
        gemm_gpu(1, 0, l.output_c, l.input_c, l.output_c, l.input_w, 1,
             l.kernel_weights, delta_n, delta_l);
    }
    update_connect_layer_gpu(l, rate, num, n_delta);
}

void update_connect_layer_gpu(Layer l, float rate, int num, float *n_delta)
{
    for (int i = 0; i < num; ++i)
    {
        int offset_i = i * l.inputs;
        int offset_o = i * l.outputs;
        float *input = l.input + offset_i;
        float *delta_n = n_delta + offset_o;
        gemm_gpu(0, 1, l.output_c, l.output_w,
             l.input_c, l.input_w, 1,
             delta_n, input, l.workspace);
        saxpy_gpu(l.update_kernel_weights, l.workspace, l.output_c * l.input_c, rate, l.update_kernel_weights);
        if (l.bias)
        {
            saxpy_gpu(l.update_bias_weights, delta_n, l.outputs, rate, l.update_bias_weights);
        }
    }
}

void update_connect_layer_weights_gpu(Layer l)
{
    hipMemcpy(l.kernel_weights, l.update_kernel_weights, l.inputs*l.outputs*sizeof(float), hipMemcpyDeviceToDevice);
    if (l.bias){
        hipMemcpy(l.bias_weights, l.update_bias_weights, l.outputs*sizeof(float), hipMemcpyDeviceToDevice);
    }
}

void save_connect_layer_weights_gpu(Layer l, FILE *fp)
{
    float *kernel_weights = (float*)calloc(l.inputs*l.outputs, sizeof(float));
    hipMemcpy(kernel_weights, l.kernel_weights, l.inputs*l.outputs*sizeof(float), hipMemcpyDeviceToHost);
    fwrite(kernel_weights, sizeof(float), l.inputs*l.outputs, fp);
    free(kernel_weights);
    if (l.bias){
        float *bias_weights = (float*)calloc(l.outputs, sizeof(float));
        hipMemcpy(bias_weights, l.bias_weights, l.outputs*sizeof(float), hipMemcpyDeviceToHost);
        fwrite(bias_weights, sizeof(float), l.outputs, fp);
        free(bias_weights);
    }
}

void free_connect_layer_gpu(Layer l)
{
    hipFree(l.output);
    hipFree(l.delta);
    hipFree(l.kernel_weights);
    hipFree(l.update_kernel_weights);
    if (l.bias){
        hipFree(l.bias_weights);
        hipFree(l.update_bias_weights);
    }
}
