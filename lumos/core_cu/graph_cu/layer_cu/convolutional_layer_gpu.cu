#include "hip/hip_runtime.h"
#include "convolutional_layer_gpu.h"

void init_convolutional_layer_gpu(Layer *l, int w, int h, int c, int subdivision)
{
    l->input_h = h;
    l->input_w = w;
    l->input_c = c;
    l->inputs = l->input_h * l->input_w * l->input_c;

    l->output_h = (l->input_h + 2 * l->pad - l->ksize) / l->stride + 1;
    l->output_w = (l->input_w + 2 * l->pad - l->ksize) / l->stride + 1;
    l->output_c = l->filters;
    l->outputs = l->output_h * l->output_w * l->output_c;

    l->workspace_size = l->ksize * l->ksize * l->input_c * l->output_h * l->output_w + l->filters * l->ksize * l->ksize * l->input_c;

    hipMalloc((void**)&l->output, subdivision*l->outputs*sizeof(float));
    hipMalloc((void**)&l->delta, subdivision*l->inputs*sizeof(float));
    hipMalloc((void**)&l->kernel_weights, l->filters*l->ksize*l->ksize*l->input_c*sizeof(float));
    hipMalloc((void**)&l->update_kernel_weights, l->filters*l->ksize*l->ksize*l->input_c*sizeof(float));
    if (l->bias){
        hipMalloc((void**)&l->bias_weights, l->filters*sizeof(float));
        hipMalloc((void**)&l->update_bias_weights, l->filters*sizeof(float));
    }

    fprintf(stderr, "Convolutional   Layer    %3d*%3d*%3d ==> %3d*%3d*%3d\n",
            l->input_w, l->input_h, l->input_c, l->output_w, l->output_h, l->output_c);
}

void weightinit_convolutional_layer_gpu(Layer l)
{
    float *kernel_weights = (float*)calloc(l.filters*l.ksize*l.ksize*l.input_c, sizeof(float));
    float *bias_weights = NULL;
    float scale = sqrt((float)2 / (l.ksize*l.ksize*l.input_c));
    for (int i = 0; i < l.filters; ++i){
        float *weight = kernel_weights + i*l.input_c*l.ksize*l.ksize;
        for (int j = 0; j < l.ksize*l.ksize; ++j){
            weight[j] = scale*rand_normal();
        }
        for (int j = 0; j < l.input_c-1; ++j){
            float *weight_c = weight + (j+1)*l.ksize*l.ksize;
            memcpy(weight_c, weight, l.ksize*l.ksize*sizeof(float));
        }
    }
    if (l.bias){
        bias_weights = (float*)calloc(l.filters, sizeof(float));
        fill_cpu(bias_weights, l.filters, 0.001, 1);
        hipMemcpy(l.bias_weights, bias_weights, l.filters*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(l.update_bias_weights, bias_weights, l.filters*sizeof(float), hipMemcpyHostToDevice);
        free(bias_weights);
    }
    hipMemcpy(l.kernel_weights, kernel_weights, l.filters*l.ksize*l.ksize*l.input_c*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(l.update_kernel_weights, kernel_weights, l.filters*l.ksize*l.ksize*l.input_c*sizeof(float), hipMemcpyHostToDevice);
    free(kernel_weights);
}

void forward_convolutional_layer_gpu(Layer l, int num)
{
    for (int i = 0; i < num; ++i){
        int offset_i = i * l.inputs;
        int offset_o = i * l.outputs;
        float *input = l.input + offset_i;
        float *output = l.output + offset_o;
        im2col_gpu(input, l.input_h, l.input_w, l.input_c, l.ksize, l.stride, l.pad, l.workspace);
        gemm_gpu(0, 0, l.filters, l.ksize * l.ksize * l.input_c, l.ksize * l.ksize * l.input_c, l.output_h * l.output_w, 1,
             l.kernel_weights, l.workspace, output);
        if (l.bias){
            add_bias_gpu(output, l.bias_weights, l.filters, l.output_h * l.output_w);
        }
        activate_list_gpu(output, l.outputs, l.active);
    }
}

void backward_convolutional_layer_gpu(Layer l, float rate, int num, float *n_delta)
{
    for (int i = 0; i < num; ++i){
        int offset_i = i * l.inputs;
        int offset_o = i * l.outputs;
        float *output = l.output + offset_o;
        float *delta_l = l.delta + offset_i;
        float *delta_n = n_delta + offset_o;
        gradient_list_gpu(output, l.outputs, l.active);
        matrix_multiply_gpu(delta_n, output, l.outputs, delta_n);
        gemm_gpu(1, 0, l.filters, l.ksize * l.ksize * l.input_c,
             l.filters, l.output_h * l.output_w, 1,
             l.kernel_weights, delta_n, l.workspace);
        col2im_gpu(l.workspace, l.ksize, l.stride, l.pad, l.input_h, l.input_w, l.input_c, delta_l);
    }
    update_convolutional_layer_gpu(l, rate, num, n_delta);
}

void update_convolutional_layer_gpu(Layer l, float rate, int num, float *n_delta)
{
    for (int i = 0; i < num; ++i){
        int offset_i = i * l.inputs;
        int offset_o = i * l.outputs;
        float *input = l.input + offset_i;
        float *delta_n = n_delta + offset_o;
        im2col_gpu(input, l.input_h, l.input_w, l.input_c, l.ksize, l.stride, l.pad, l.workspace);
        gemm_gpu(0, 1, l.filters, l.output_h * l.output_w,
             l.ksize * l.ksize * l.input_c, l.output_h * l.output_w, 1,
             delta_n, l.workspace, l.workspace + l.ksize * l.ksize * l.input_c * l.output_h * l.output_w);
        saxpy_gpu(l.update_kernel_weights, l.workspace + l.ksize * l.ksize * l.input_c * l.output_h * l.output_w, l.filters * l.ksize * l.ksize * l.input_c, rate, l.update_kernel_weights);
        if (l.bias){
            sum_channel_gpu(delta_n, l.output_h, l.output_w, l.output_c, rate, l.workspace);
            add_bias_gpu(l.update_bias_weights, l.workspace, l.output_c, 1);
        }
    }
}

void update_convolutional_layer_weights_gpu(Layer l)
{
    hipMemcpy(l.kernel_weights, l.update_kernel_weights, l.filters*l.ksize*l.ksize*l.input_c*sizeof(float), hipMemcpyDeviceToDevice);
    if (l.bias){
        hipMemcpy(l.bias_weights, l.update_bias_weights, l.filters*sizeof(float), hipMemcpyDeviceToDevice);
    }
}
