#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

#include "gpu.h"

__global__ void fill_kernel(float *data, int len, float x, int offset)
{
    int index = (blockDim.x * blockIdx.x + threadIdx.x)*offset;
    if (index >= len) return;
    data[index] = x;
}

void fill_gpu(float *data, int len, float x, int offset)
{
    fill_kernel<<<(len+BLOCK-1)/BLOCK, BLOCK>>>(data, len, x, offset);
}

void multy_gpu(float *data, int len, float x, int offset);
// void add_gpu(float *data, int len, float x, int offset);

float min_gpu(float *data, int num);
float max_gpu(float *data, int num);
float sum_gpu(float *data, int num);
float mean_gpu(float *data, int num);

void add_gpu(float *data_a, float *data_b, int num, float *space);
void subtract_gpu(float *data_a, float *data_b, int num, float *space);
void multiply_gpu(float *data_a, float *data_b, int num, float *space);
void divide_gpu(float *data_a, float *data_b, int num, float *space);

void saxpy_gpu(float *data_a, float *data_b, int num, float x, float *space);
