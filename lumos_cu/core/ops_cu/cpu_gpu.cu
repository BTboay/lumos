#include "hip/hip_runtime.h"
#include "cpu_gpu.h"

__global__ void fill_kernel(float *data, int len, float x, int offset)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x)*offset;
    if (index >= len) return;
    data[index] = x;
}

__global__ void multy_kernel(float *data, int len, float x, int offset)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x)*offset;
    if (index >= len) return;
    data[index] *= x;
}

__global__ void add_kernel(float *data, int len, float x, int offset)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x)*offset;
    if (index >= len) return;
    data[index] += x;
}

void fill_gpu(float *data, int len, float x, int offset)
{
    fill_kernel<<<(len+BLOCK-1)/BLOCK, BLOCK>>>(data, len, x, offset);
}

void multy_gpu(float *data, int len, float x, int offset)
{
    multy_kernel<<<(len+BLOCK-1)/BLOCK, BLOCK>>>(data, len, x, offset);
}

void add_gpu(float *data, int len, float x, int offset)
{
    add_kernel<<<(len+BLOCK-1)/BLOCK, BLOCK>>>(data, len, x, offset);
}

__global__ void matrix_add_kernel(float *data_a, float *data_b, int num, float *space)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= num) return;
    space[index] = data_a[index] + data_b[index];
}

__global__ void matrix_subtract_kernel(float *data_a, float *data_b, int num, float *space)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= num) return;
    space[index] = data_a[index] - data_b[index];
}

__global__ void matrix_multiply_kernel(float *data_a, float *data_b, int num, float *space)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= num) return;
    space[index] = data_a[index] * data_b[index];
}

__global__ void matrix_divide_kernel(float *data_a, float *data_b, int num, float *space)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= num) return;
    space[index] = data_a[index] / data_b[index];
}

__global__ void saxpy_kernel(float *data_a, float *data_b, int num, float x, float *space)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= num) return;
    space[index] = data_a[index] + x * data_b[index];
}

void matrix_add_gpu(float *data_a, float *data_b, int num, float *space)
{
    matrix_add_kernel<<<(num+BLOCK-1)/BLOCK, BLOCK>>>(data_a, data_b, num, space);
}

void matrix_subtract_gpu(float *data_a, float *data_b, int num, float *space)
{
    matrix_subtract_kernel<<<(num+BLOCK-1)/BLOCK, BLOCK>>>(data_a, data_b, num, space);
}

void matrix_multiply_gpu(float *data_a, float *data_b, int num, float *space)
{
    matrix_multiply_kernel<<<(num+BLOCK-1)/BLOCK, BLOCK>>>(data_a, data_b, num, space);
}

void matrix_divide_gpu(float *data_a, float *data_b, int num, float *space)
{
    matrix_divide_kernel<<<(num+BLOCK-1)/BLOCK, BLOCK>>>(data_a, data_b, num, space);
}

void saxpy_gpu(float *data_a, float *data_b, int num, float x, float *space)
{
    saxpy_kernel<<<(num+BLOCK-1)/BLOCK, BLOCK>>>(data_a, data_b, num, x, space);
}
