#include "im2col_layer_gpu.h"

void forward_im2col_layer(Layer l, int num)
{
    hipMemcpy(l.output, l.input, num*l.outputs*sizeof(float), hipMemcpyDeviceToDevice);
}

void backward_im2col_layer(Layer l, float rate, int num, float *n_delta)
{
    hipMemcpy(l.delta, n_delta, num*l.inputs*sizeof(float), hipMemcpyDeviceToDevice);
}
